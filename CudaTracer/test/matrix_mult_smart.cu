#include "hip/hip_runtime.h"
#include "CustomAllocatorManager.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

constexpr int NUM = 1024;
constexpr int ITERS = 512;
constexpr int matrix_size = NUM * NUM * sizeof(double);

__global__ void matMul(const double *A, const double *B, double *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Row index of C to compute
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Column index of C to compute

    if(row < NUM && col < NUM){
        double value = 0.0f;
        for(int k = 0; k < NUM; k++){
            value += A[row * NUM + k] * B[k * NUM + col];
        }
        C[row * NUM + col] = value;
    }
}


void saveFinalResult(double* matrix) {
    // Calculate the checksum of the matrix
    double checksum = 0.0f;
    for (int i = 0; i < NUM * NUM; i++) {
        checksum += matrix[i];
    }
    std::cout << "Checksum of the final result: " << checksum << std::endl;
}

void updateIntermediateResult(double* matrix) {
    matrix[0] += 0.1f;
}


void perform_test() {
    double *d_A, *d_B, *d_C, *d_D, *d_E;
    double *h_A, *h_B, *h_C, *h_D, *h_E;

    // Device-side memory allocation (ommits error checking)
    hipMalloc((void**)&d_A, matrix_size);
    hipMalloc((void**)&d_B, matrix_size);
    hipMalloc((void**)&d_C, matrix_size);
    hipMalloc((void**)&d_D, matrix_size);
    hipMalloc((void**)&d_E, matrix_size);

    // Host-side memory allocation using custom allocator
    h_A = static_cast<double *>(g_allocator_manager.allocate_memory(matrix_size));
    h_B = static_cast<double *>(g_allocator_manager.allocate_memory(matrix_size));
    h_C = static_cast<double *>(g_allocator_manager.allocate_memory(matrix_size));
    h_D = static_cast<double *>(g_allocator_manager.allocate_memory(matrix_size));
    h_E = static_cast<double *>(g_allocator_manager.allocate_memory(matrix_size));

    
    // Initialize matrices with 1s for h_A, h_B, and h_D
    for(int i = 0; i < NUM; i++){
        for(int j = 0; j < NUM; j++){
            h_A[i * NUM + j] = 1.0;
            h_B[i * NUM + j] = 1.0;
            h_D[i * NUM + j] = 1.0;
        }
    }

    // Synchronize to ensure timing is accurate
    hipDeviceSynchronize();
    auto start = std::chrono::high_resolution_clock::now();

    // Transfer from unpinned memory to the device (slow)
    hipMemcpy(d_A, h_A, matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrix_size, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 blockDim(32, 32); // 32x32 threads per block
    dim3 gridDim((NUM + blockDim.x - 1) / blockDim.x, (NUM + blockDim.y - 1) / blockDim.y); // Grid size to cover the entire matrix

    // Perform first operation, C = A x B
    matMul<<<gridDim, blockDim>>>(d_A, d_B, d_C);

    // Transfer result back (assuming intermediate results modified by processIntermediateResult)
    hipMemcpy(h_C, d_C, matrix_size, hipMemcpyDeviceToHost);
    updateIntermediateResult(h_C);

    // Transfer matrix D to device 
    hipMemcpy(d_D, h_D, matrix_size, hipMemcpyHostToDevice);
    for (int i = 0; i < ITERS; i++) {
        
        // Transfer matrix C to devce
        hipMemcpy(d_C, h_C, matrix_size, hipMemcpyHostToDevice);

        // Perform repeat operation, E = C x D
        matMul<<<gridDim, blockDim>>>(d_C, d_D, d_E);

        // Move result from d_E back to h_C, save result, and repeat
        if (i != (ITERS - 1)) {
            hipMemcpy(h_C, d_E, matrix_size, hipMemcpyDeviceToHost);
            updateIntermediateResult(h_C);
        }
    }

    // Move final result 
    hipMemcpy(h_E, d_E, matrix_size, hipMemcpyDeviceToHost);

    // Synchronize and end timing
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate and print the duration
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    std::cout << "Duration: " << duration.count() << " ms" << std::endl;

    saveFinalResult(h_E);

    // Clean up device memory 
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);
    hipFree(d_E);

    // Clean up host memory
    g_allocator_manager.deallocate_memory(h_A, matrix_size);
    g_allocator_manager.deallocate_memory(h_B, matrix_size);
    g_allocator_manager.deallocate_memory(h_C, matrix_size);
    g_allocator_manager.deallocate_memory(h_D, matrix_size);
    g_allocator_manager.deallocate_memory(h_E, matrix_size);
}


int main() {

    // First run in profile mode to generate the history file
    g_allocator_manager.initialize("profile"); 
    perform_test();

    // Run in use mode
    g_allocator_manager.initialize("use", false);
    perform_test();

}