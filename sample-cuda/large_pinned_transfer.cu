#include <iostream>
#include <hip/hip_runtime.h>
#include <unistd.h> // For getpid()
#include <sys/mman.h>

#define PAGE_SIZE 4096

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void waitForUser() {
    pid_t pid = getpid();
    std::cout << "Process ID (PID): " << pid << std::endl;
    std::cout << "Press Enter to continue..." << std::endl;
    std::cin.get();
}


void largeTransfer(size_t size) {
    // Use CudaHostAlloc to allocate pinned memory
    char* h_src = nullptr;
    char* h_dst = nullptr;

    std::cout << "[largeTransfer] Allocating " << size << " bytes of pinned memory." << std::endl;
    waitForUser();

   checkCudaError(hipHostAlloc(&h_src, size, hipHostMallocDefault), "hipHostAlloc h_src");
    // h_src = (char*)mmap(nullptr, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS, -1, 0);
    // if (h_src == MAP_FAILED) {
    //     std::cerr << "Error: Unable to allocate host memory using mmap." << std::endl;
    //     exit(EXIT_FAILURE);
    // }
    // Wait for user input
    std::cout << "[largeTransfer] memory allocated but not used" << std::endl;
    waitForUser();

    // Touch each page to ensure it is allocated
    for (size_t i = 0; i < size; i += PAGE_SIZE) {
        h_src[i] = 0x52;
    }

    // Wait for user input
    std::cout << "[largeTransfer] memory allocated and used" << std::endl;
    waitForUser();

    // Transfer data from host to device
    char* d_src = nullptr;
    checkCudaError(hipMalloc(&d_src, size), "hipMalloc d_src");
    checkCudaError(hipMemcpy(d_src, h_src, size, hipMemcpyHostToDevice), "hipMemcpy h_src to d_src");

    // Wait for user input
    std::cout << "[largeTransfer] data transferred to device, ready to free" << std::endl;
    waitForUser();

    // Free memory
    checkCudaError(hipFree(d_src), "hipFree d_src");
    checkCudaError(hipHostFree(h_src), "hipHostFree h_src");
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <size_in_MB>" << std::endl;
        return EXIT_FAILURE;
    }

    size_t sizeInMB = std::stoul(argv[1]);
    size_t sizeInBytes = sizeInMB * 1024 * 1024;

    largeTransfer(sizeInBytes);

    std::cout << "Program completed successfully." << std::endl;
    return 0;
}
